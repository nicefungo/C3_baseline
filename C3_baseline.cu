#include "hip/hip_runtime.h"
#include "C3_baseline.h"
#include "util.h"

void TODO(){
	std::cout << "TODO stuff undone in `cuda_baseline.cu`." << std::endl;
	std::exit(EXIT_FAILURE);
}

template<typename T>
__global__ void fused_3200x16x32_3200x16x16_SiLU(const T * input, const T * \
                Conv1_weight, const T * Conv1_bias, const T * Convm0_weight,\
                const T * Convm0_bias, T * D1, T * D2, unsigned int offset)
{
	//TODO
	return;
}

template<typename T>
__global__ void Convm1_trivial(const T * input, const T * weight, const T * \
                bias, T * D, unsigned int offset)
{
	TODO();
}



template<typename T>
void CPU_Conv_3200x16x32_SiLU(const T * input, const T * weight, const T * bias, \
		T * D)
{
	for (int i = 0; i < 3200 * 16; ++i) {
        	D[i] = 0;
    	}
	for(int m = 0; m < 3200; m++){
		for(int n = 0; n < 16; n++){
			for(int k = 0; k < 32; k++){
				D[m * 16 + n] += input[m * 32 + k] \
						 * weight[k * 16 + n];
			}
			
			D[m * 16 + n] += bias[n];
			// TODO: SiLU
			D[m * 16 + n] = silu<float>(D[m * 16 + n]);
			
		}

	}

	return;
}


template<typename T>
__global__ void Conv_3200x16x32_SiLU(const T * input, const T * weight, \
                const T * bias, T * D, unsigned int offset)
{

	// Input is a matrix transposed from img
	//
	// Matrix size: 25600 * 32
	//
	// Multiplying of a single img is divided into 8
	// asyn processes on separate streams indexed by
	// offset

	// GEMM
	// -----------------------------------------------
	// M = 3200, N = 16, K = 32
	// Tiling size = M: 16, N: 16, K: 16
	// TODO: try tiling size M: 16, N: 16, K: 8
	//                    or M: 32, N: 16, K: 8
	//                    or M: 32, N: 16, K: 16
	//                    or M: 64, N: 16, K: 16
	// Tiling dim: tild_M = 100, tile_N = 2, tile_K = 1
	

	// default block size 16 * 16
	// each block tile compute a 16 * 16 tile

	unsigned int start_pos = (offset * 3200U) << 5;

	unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;

	// block tile size 16 * 16
	// block size 16 * 16
	unsigned int block_linear = gridDim.x * blockIdx.y + blockIdx.x;
	unsigned int thread_linear = blockDim.x * threadIdx.y + threadIdx.x;

	// 8 warps per block
	// unsigned int warp_id = (thread_linear >> 5);
	// unsigned int lane_id = thread_linear & 31U;
	
	
	// Totally 512 * 2 elements to move for each thread in 
	// block
	//
	// Two elements to each shared mem
	//
	// Do padding on the column to avoid shared mem access
	// conflict


	// Get everything required from matrix in register
	// Two accesses to global mem : Global -> Shared
	// 

	__shared__ T tiled_input[16][33];
	// __shared__ T tiled_input_transposed[32][17]
	
	// An offset of thread_linear or (* + 256) on tiled input
	tiled_input[thread_linear >> 5][thread_linear & 31U] = input[start_pos
                                                    + (blockIdx.y << 9)
                                                    + thread_linear];
        tiled_input[(thread_linear >> 5) + 8][thread_linear & 31U]
						            = input[start_pos
                                                    + (blockIdx.y << 9)
                                                    + thread_linear + 256U];

	/*
	tiled_input_transposed[threadIdx.y][threadIdx.x] = input[start_pos 
						    + block_linear * 256
						    + thread_linear];
	tiled_input_transposed[threadIdx.y][threadIdx.x + 16] = input[start_pos 
						    + block_linear * 256
						    + thread_linear + 256];
	*/
	
	
	__shared__ T tiled_weight[32][17];
	// Coalescing access
	tiled_weight[threadIdx.y][threadIdx.x] = weight[thread_linear];
	tiled_weight[threadIdx.y + 16U][threadIdx.x] = weight[thread_linear + 256U];

	__syncthreads();

	// 
	// First trying a warp tiling of size 4 * 2
	
	// No warp tiling for now
	// unsigned int warp_tile_id_x = warp_id >> 1;
	// unsigned int warp_tile_id_y = warp_id & 1;
	
	T sum{static_cast<T>(0)};

#pragma unroll
	for(int i = 0; i < 32; i++){
		sum += tiled_weight[i][threadIdx.x] * tiled_input[threadIdx.y][i];
	}
	
	sum += bias[col];

	sum = silu<T>(sum);

	// TODO: SiLU;


	D[(start_pos >> 1) + (row << 4)+ col] = sum;

	return;
}



template<typename T>
__global__ void Conv_3200x32x32_SiLU(const T * input, const T * weight, \
                const T * bias, T * D, unsigned int offset)
{
	// 
}

template<typename T>
void C3(const T * input, const T * weights, const T * biases, T * D, T * buffer){
	
	// kernel size dummy
	int gridsize_dummy = 1, blocksize_dummy = 1;

	// Part_1
	hipStream_t streams1[8];
	hipStream_t streams2[8];
	//hipStream_t streams3[16];
	//hipStream_t streams4[16];

	for(int i = 0; i < 8; i++){
		hipStreamCreate(&streams1[i]);
	}

	for(int i = 0; i < 8; i++){
		fused_3200x16x32_3200x16x16_SiLU<T>
			<<<gridsize_dummy, blocksize_dummy, 0, streams1[i]>>>
			(input,
		 	weights + CONV_WEIGHT_1_OFFSET, 
		 	biases + CONV_BIAS_1_OFFSET, 
		 	weights + CONV_WEIGHT_m0_OFFSET, 
		 	biases + CONV_BIAS_m0_OFFSET, 
		 	D,
			buffer,
			i
		 	);
	}

	for(int i = 0; i < 8; i++){
		hipStreamCreate(&streams2[i]);
	}

	for(int i = 0; i < 8; i++){
		std::cout << i << "-th iteration of Conv1 beginning." << std::endl;
		Conv_3200x16x32_SiLU<T>
			<<<gridsize_dummy, blocksize_dummy, 0, streams2[i]>>>
			(input,
			 weights + CONV_WEIGHT_2_OFFSET,
			 biases + CONV_BIAS_2_OFFSET,
			 buffer,
			 i
			 );
	}

	// Temp code for freeing resources in main
	// TODO: remove after finishing the following part
	for(int i = 0; i < 8; i++){
		CHECK_CUDA_ERROR(hipStreamSynchronize(streams2[i]));
		std::cout << i << "-th iteration of Conv1 finished." << std::endl;
		CHECK_CUDA_ERROR(hipStreamDestroy(streams2[i]));
	}

	for(int i = 0; i < 8; i++){
		CHECK_CUDA_ERROR(hipStreamSynchronize(streams1[i]));
		CHECK_CUDA_ERROR(hipStreamDestroy(streams1[i]));
	}

	// Part_2
}

int main(int arg, char ** args){
	bool test_flags[5];
	if(arg>=2){
		for(int i = 0; i < 5; i++){
			test_flags[i] = (args[1][i]=='1');
		}
	
	}

	float * input, * weights, * biases, * output, * buffer;
	CHECK_CUDA_ERROR(hipHostAlloc((void**)&input, INPUT_SIZE * sizeof(float), hipHostMallocDefault));
	CHECK_CUDA_ERROR(hipHostAlloc((void**)&weights, CONV_WEIGHT_SIZE * sizeof(float), hipHostMallocDefault));
	CHECK_CUDA_ERROR(hipHostAlloc((void**)&biases, CONV_BIAS_SIZE * sizeof(float), hipHostMallocDefault));
	CHECK_CUDA_ERROR(hipHostAlloc((void**)&output, OUTPUT_SIZE * sizeof(float), hipHostMallocDefault));

	// Use pinned mem instead
	// float * input = (float *)malloc(INPUT_SIZE * sizeof(float));
	// float * weights = (float *)malloc(CONV_WEIGHT_SIZE * sizeof(float));
	// float * biases = (float *)malloc(CONV_BIAS_SIZE * sizeof(float));
	// float * output = (float *)malloc(OUTPUT_SIZE * sizeof(float));

	// Temp and Output utilization;
	// Input --Conv1--> [Temp + 0u] --Convm0,Convm1--> [Temp + OUTPUT_SIZE >> 1]
	//                            \-------------------------------\ --Add--
        //                                             [Output + 0u] <--Move--/
	// Input --Conv2--> [Output + OUTPUT_SIZE >> 1]

	
	// Use pinned mem instead
	// float * buffer = (float *)malloc(OUTPUT_SIZE * sizeof(float)); 
	CHECK_CUDA_ERROR(hipHostAlloc((void**)&buffer, OUTPUT_SIZE * sizeof(float), hipHostMallocDefault));


	float * d_input, * d_weights, * d_biases, * d_output, * d_buffer;
	CHECK_CUDA_ERROR(hipMalloc((void**)&d_input, INPUT_SIZE * sizeof(float)));
	CHECK_CUDA_ERROR(hipMalloc((void**)&d_weights, CONV_WEIGHT_SIZE * sizeof(float)));
	CHECK_CUDA_ERROR(hipMalloc((void**)&d_biases, CONV_BIAS_SIZE * sizeof(float)));
	CHECK_CUDA_ERROR(hipMalloc((void**)&d_output, OUTPUT_SIZE * sizeof(float)));
	CHECK_CUDA_ERROR(hipMalloc((void**)&d_buffer, OUTPUT_SIZE * sizeof(float)));


	

		// Unit tests
		std::cout << "Unit tests begin" << std::endl
                	<< "-----------------------------------------------"
                	<< std::endl
                	<< "-----------------------------------------------"
                	<< std::endl
			<< std::endl;
	if(test_flags[0]){	
		// Test for Conv 3200 * 32 * 32 with Gelu activation
		 
		std::cout << "Unit Test 1 on Conv3 begins." << std::endl;
		std::cout << "-----------------------------------------------"
			  << std::endl;

		std::cout << "Unit Test 1 on Conv3 done." << std::endl
                        << std::endl;
		
	}

	if(test_flags[1]){
		// Test for Conv 3200 * 16 * 32 with Gelu activation
		std::cout << "Unit Test 2 on Conv1 begins." << std::endl;
		std::cout << "-----------------------------------------------"
			  << std::endl;

		float * t_input_2, * t_weight_2, * t_bias_2, * t_output_2;

		CHECK_CUDA_ERROR(hipHostAlloc((void**)&t_input_2, 100U * sizeof(float) << 10, hipHostMallocDefault));
		CHECK_CUDA_ERROR(hipHostAlloc((void**)&t_weight_2, sizeof(float) << 9, hipHostMallocDefault));
		CHECK_CUDA_ERROR(hipHostAlloc((void**)&t_bias_2, sizeof(float) << 4, hipHostMallocDefault));

		

		CHECK_CUDA_ERROR(hipHostAlloc((void**)&t_output_2, 100U * sizeof(float) << 9, hipHostMallocDefault));
		float * t_gt_2 = (float *)malloc(100U * sizeof(float) << 9);
		memset((void*) t_gt_2, 0, 100U * sizeof(float) << 9);

		for(int i = 0; i < 102400; i++){
			t_input_2[i] = static_cast<float>(rand()) \
				       / static_cast<float>(RAND_MAX) - 0.5f;
		}

		for(int i = 0; i < 512; i++){
			t_weight_2[i] = static_cast<float>(rand()) \
                                       / static_cast<float>(RAND_MAX) - 0.5f;
		}

		for(int i = 0; i < 16; i++){
                        t_bias_2[i] = static_cast<float>(rand()) \
                                       / static_cast<float>(RAND_MAX);
                }

		float * d_t_input_2, * d_t_weight_2, * d_t_bias_2, * d_t_output_2;
		CHECK_CUDA_ERROR(hipMalloc((void**)&d_t_input_2, 100U * sizeof(float) << 10));
		CHECK_CUDA_ERROR(hipMalloc((void**)&d_t_weight_2, sizeof(float) << 9));
		CHECK_CUDA_ERROR(hipMalloc((void**)&d_t_bias_2, sizeof(float) << 4));
		CHECK_CUDA_ERROR(hipMalloc((void**)&d_t_output_2, 100U * sizeof(float) << 9));
			
		CHECK_CUDA_ERROR(hipMemcpy(d_t_input_2, t_input_2, 100U * \ 
					sizeof(float) << 10, hipMemcpyHostToDevice));
		CHECK_CUDA_ERROR(hipMemcpy(d_t_weight_2, t_weight_2, \
					sizeof(float) << 9,  hipMemcpyHostToDevice));
		CHECK_CUDA_ERROR(hipMemcpy(d_t_bias_2, t_bias_2, \ 
                                        sizeof(float) << 4, hipMemcpyHostToDevice));
		CHECK_CUDA_ERROR(hipMemset((void*)d_t_output_2, 0, 100U * sizeof(float) << 9));

		dim3 block_size_2(16, 16);
		dim3 grid_size_2(1, 200);
		Conv_3200x16x32_SiLU<float>
			<<<grid_size_2, block_size_2>>>
				(d_t_input_2, \
				 d_t_weight_2, \
				 d_t_bias_2, \
			         d_t_output_2, 0);


		CHECK_LAST_CUDA_ERROR();

		CHECK_CUDA_ERROR(hipMemcpy(t_output_2, d_t_output_2, 100U * \
					sizeof(float) << 9, hipMemcpyDeviceToHost));

		CPU_Conv_3200x16x32_SiLU(t_input_2, t_weight_2, t_bias_2, t_gt_2);	

		bool flag2 = true;
		int false_num = 0;
		for(int i = 0; i < 3200; i++){
			for(int j = 0; j < 16; j++){
				if(i >= 0 && i <= 3 && j >= 0 && j <= 3){
					std::cout << t_gt_2[i * 16 + j] << " " << t_output_2[i * 16 + j] << std::endl;
				}
				if(abs(t_gt_2[i * 16 + j] - t_output_2[i * 16 + j]) > 0.001f){
					flag2 = false;
					false_num++;
				}
			}
		}
		std::cout << "Test 2 passed?: " << flag2 << std::endl;
		std::cout << "Mistake on "<< false_num << " elements out of 51200" << std::endl;

		
	       	CHECK_CUDA_ERROR(hipHostFree(t_input_2));	
	       	CHECK_CUDA_ERROR(hipHostFree(t_weight_2));	
	       	CHECK_CUDA_ERROR(hipHostFree(t_bias_2));	
	       	CHECK_CUDA_ERROR(hipHostFree(t_output_2));

	       	CHECK_CUDA_ERROR(hipFree(d_t_input_2));	
	       	CHECK_CUDA_ERROR(hipFree(d_t_weight_2));	
	       	CHECK_CUDA_ERROR(hipFree(d_t_bias_2));	
	       	CHECK_CUDA_ERROR(hipFree(d_t_output_2));

		free(t_gt_2);	
		std::cout << "Unit Test 2 on Conv1 done." << std::endl
			<< std::endl;
	}


	if(test_flags[2]){

		// Test for Fused Conv 3200 * 32 * 32 with Gelu activation
		// and Conv 3200 * 16 * 16 with Gelu activation
		std::cout << "Unit Test 3 on fused layers begins." << std::endl;
                std::cout << "-----------------------------------------------"
                          << std::endl;	

		std::cout << "Unit Test 3 on fused layers done." << std::endl
                        << std::endl;
	}

	if(test_flags[3]){

                // Test for 3x3 kernel Conv 3200 * 16 * 16 with Gelu activation
                std::cout << "Unit Test 4 on 3x3 Conv begins." << std::endl;
                std::cout << "-----------------------------------------------"
                          << std::endl;

                std::cout << "Unit Test 4 on 3x3 Conv done." << std::endl
                        << std::endl;
        }

	if(test_flags[4]){

                // Test for ? with Gelu activation
                std::cout << "Unit Test 5 on ? begins." << std::endl;
                std::cout << "-----------------------------------------------"
                          << std::endl;

                std::cout << "Unit Test 5 on ? done." << std::endl
                        << std::endl;
        }

	


	// Init or IO with ONNX exported files
	// TODO


	// TODO: finish C3 Test
	// TODO: There are unsolved illegal mem accesses in Part_1 C3
	// C3<float>(input, weights, biases, output, buffer);
	
	CHECK_LAST_CUDA_ERROR();
	CHECK_CUDA_ERROR(hipDeviceSynchronize());

	CHECK_CUDA_ERROR(hipFree(d_input));
	CHECK_CUDA_ERROR(hipFree(d_weights));
	CHECK_CUDA_ERROR(hipFree(d_biases));
	CHECK_CUDA_ERROR(hipFree(d_output));
	CHECK_CUDA_ERROR(hipFree(d_buffer));

	CHECK_CUDA_ERROR(hipHostFree(input));
	CHECK_CUDA_ERROR(hipHostFree(weights));
	CHECK_CUDA_ERROR(hipHostFree(biases));
	CHECK_CUDA_ERROR(hipHostFree(output));
	CHECK_CUDA_ERROR(hipHostFree(buffer));
	
	// Use pinned memory instead
	// free(input); free(weights); free(biases); free(output); free(buffer);
	
	CHECK_CUDA_ERROR(hipDeviceReset());
	return 0;
}

